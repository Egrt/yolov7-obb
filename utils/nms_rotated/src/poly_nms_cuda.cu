#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;


#define maxn 10
const double eps=1E-8;

__device__ inline int sig(float d){
    return(d>1E-8)-(d<-1E-8);
}

__device__ inline int point_eq(const float2 a, const float2 b) {
    return sig(a.x - b.x) == 0 && sig(a.y - b.y)==0;
}

__device__ inline void point_swap(float2 *a, float2 *b) {
    float2 temp = *a;
    *a = *b;
    *b = temp;
}

__device__ inline void point_reverse(float2 *first, float2* last)
{
    while ((first!=last)&&(first!=--last)) {
        point_swap (first,last);
        ++first;
    }
}

__device__ inline float cross(float2 o,float2 a,float2 b){  //叉积
    return(a.x-o.x)*(b.y-o.y)-(b.x-o.x)*(a.y-o.y);
}
__device__ inline float area(float2* ps,int n){
    ps[n]=ps[0];
    float res=0;
    for(int i=0;i<n;i++){
        res+=ps[i].x*ps[i+1].y-ps[i].y*ps[i+1].x;
    }
    return res/2.0;
}
__device__ inline int lineCross(float2 a,float2 b,float2 c,float2 d,float2&p){
    float s1,s2;
    s1=cross(a,b,c);
    s2=cross(a,b,d);
    if(sig(s1)==0&&sig(s2)==0) return 2;
    if(sig(s2-s1)==0) return 0;
    p.x=(c.x*s2-d.x*s1)/(s2-s1);
    p.y=(c.y*s2-d.y*s1)/(s2-s1);
    return 1;
}

__device__ inline void polygon_cut(float2*p,int&n,float2 a,float2 b, float2* pp){

    int m=0;p[n]=p[0];
    for(int i=0;i<n;i++){
        if(sig(cross(a,b,p[i]))>0) pp[m++]=p[i];
        if(sig(cross(a,b,p[i]))!=sig(cross(a,b,p[i+1])))
            lineCross(a,b,p[i],p[i+1],pp[m++]);
    }
    n=0;
    for(int i=0;i<m;i++)
        if(!i||!(point_eq(pp[i], pp[i-1])))
            p[n++]=pp[i];
    // while(n>1&&p[n-1]==p[0])n--;
    while(n>1&&point_eq(p[n-1], p[0]))n--;
}

//---------------华丽的分隔线-----------------//
//返回三角形oab和三角形ocd的有向交面积,o是原点//
__device__ inline float intersectArea(float2 a,float2 b,float2 c,float2 d){
    float2 o = make_float2(0,0);
    int s1=sig(cross(o,a,b));
    int s2=sig(cross(o,c,d));
    if(s1==0||s2==0)return 0.0;//退化，面积为0
    // if(s1==-1) swap(a,b);
    // if(s2==-1) swap(c,d);
    if (s1 == -1) point_swap(&a, &b);
    if (s2 == -1) point_swap(&c, &d);
    float2 p[10]={o,a,b};
    int n=3;
    float2 pp[maxn];
    polygon_cut(p,n,o,c,pp);
    polygon_cut(p,n,c,d,pp);
    polygon_cut(p,n,d,o,pp);
    float res=fabs(area(p,n));
    if(s1*s2==-1) res=-res;return res;
}
//求两多边形的交面积
__device__ inline float intersectArea(float2*ps1,int n1,float2*ps2,int n2){
    if(area(ps1,n1)<0) point_reverse(ps1,ps1+n1);
    if(area(ps2,n2)<0) point_reverse(ps2,ps2+n2);
    ps1[n1]=ps1[0];
    ps2[n2]=ps2[0];
    float res=0;
    for(int i=0;i<n1;i++){
        for(int j=0;j<n2;j++){
            res+=intersectArea(ps1[i],ps1[i+1],ps2[j],ps2[j+1]);
        }
    }
    return res;//assumeresispositive!
}

// TODO: optimal if by first calculate the iou between two hbbs
__device__ inline float devPolyIoU(float const * const p, float const * const q) {
    float2 ps1[maxn], ps2[maxn];
    int n1 = 4;
    int n2 = 4;
    for (int i = 0; i < 4; i++) {
        ps1[i].x = p[i * 2];
        ps1[i].y = p[i * 2 + 1];

        ps2[i].x = q[i * 2];
        ps2[i].y = q[i * 2 + 1];
    }
    float inter_area = intersectArea(ps1, n1, ps2, n2);
    float union_area = fabs(area(ps1, n1)) + fabs(area(ps2, n2)) - inter_area;
    float iou = 0;
    if (union_area == 0) {
        iou = (inter_area + 1) / (union_area + 1);
    } else {
        iou = inter_area / union_area;
    }
    return iou;
}

__global__ void poly_nms_kernel(const int n_polys, const float nms_overlap_thresh,
                            const float *dev_polys, unsigned long long *dev_mask) {
    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    const int row_size =
            min(n_polys - row_start * threadsPerBlock, threadsPerBlock);
    const int cols_size =
            min(n_polys - col_start * threadsPerBlock, threadsPerBlock);

    __shared__ float block_polys[threadsPerBlock * 9];
    if (threadIdx.x < cols_size) {
        block_polys[threadIdx.x * 9 + 0] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 0];
        block_polys[threadIdx.x * 9 + 1] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 1];
        block_polys[threadIdx.x * 9 + 2] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 2];
        block_polys[threadIdx.x * 9 + 3] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 3];
        block_polys[threadIdx.x * 9 + 4] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 4];
        block_polys[threadIdx.x * 9 + 5] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 5];
        block_polys[threadIdx.x * 9 + 6] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 6];
        block_polys[threadIdx.x * 9 + 7] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 7];
        block_polys[threadIdx.x * 9 + 8] =
            dev_polys[(threadsPerBlock * col_start + threadIdx.x) * 9 + 8];
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const float *cur_box = dev_polys + cur_box_idx * 9;
        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
            start = threadIdx.x + 1;
        }
        for (i = start; i < cols_size; i++) {
            if (devPolyIoU(cur_box, block_polys + i * 9) > nms_overlap_thresh) {
                t |= 1ULL << i;
            }
        }
        const int col_blocks = THCCeilDiv(n_polys, threadsPerBlock);
        dev_mask[cur_box_idx * col_blocks + col_start] = t;
    }
}

// boxes is a N x 9 tensor
at::Tensor poly_nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {

    at::DeviceGuard guard(boxes.device());

    using scalar_t = float;
    AT_ASSERTM(boxes.device().is_cuda(), "boxes must be a CUDA tensor");
    auto scores = boxes.select(1, 8);
    auto order_t = std::get<1>(scores.sort(0, /*descending=*/true));
    auto boxes_sorted = boxes.index_select(0, order_t);

    int boxes_num = boxes.size(0);

    const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

    scalar_t* boxes_dev = boxes_sorted.data_ptr<scalar_t>();

    THCState *state = at::globalContext().lazyInitCUDA();

    unsigned long long* mask_dev = NULL;

    mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

    dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
                THCCeilDiv(boxes_num, threadsPerBlock));
    dim3 threads(threadsPerBlock);
    poly_nms_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(boxes_num,
                                        nms_overlap_thresh,
                                        boxes_dev,
                                        mask_dev);
    
    std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
    THCudaCheck(hipMemcpyAsync(
			    &mask_host[0],
                            mask_dev,
                            sizeof(unsigned long long) * boxes_num * col_blocks,
                            hipMemcpyDeviceToHost,
			    at::cuda::getCurrentCUDAStream()
			    ));
    
    std::vector<unsigned long long> remv(col_blocks);
    memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

    at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
    int64_t* keep_out = keep.data_ptr<int64_t>();

    int num_to_keep = 0;
    for (int i = 0; i < boxes_num; i++) {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;

        if (!(remv[nblock] & (1ULL << inblock))) {
            keep_out[num_to_keep++] = i;
            unsigned long long *p = &mask_host[0] + i * col_blocks;
            for (int j = nblock; j < col_blocks; j++) {
                remv[j] |= p[j];
            }
        }
    }

    THCudaFree(state, mask_dev);

    return order_t.index({
        keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
          order_t.device(), keep.scalar_type())});
}

